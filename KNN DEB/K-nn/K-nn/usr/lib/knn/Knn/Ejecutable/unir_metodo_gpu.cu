#include <stdio.h>
#include <string.h>

int system(const char *command);

int return_major(hipDeviceProp_t devProp)
{
	return devProp.major;
}
int return_minor(hipDeviceProp_t devProp)
{
	return devProp.minor;
}
int major, minor;
int main(int argc, char *argv[])
{

	int devCount;
	hipGetDeviceCount(&devCount);


    // Iterate through devices
	for (int i = 0; i < devCount; ++i)
	{
        // Get device properties
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		printDevProp(devProp);

		major = return_major(devProp);
		minor = return_minor(devProp);

	}

	if (argc != 5) {
		printf("\nError :: Ejecutar como : salida.out file1 file2 file3\n");
		return 0;
	}

	FILE *p, *q, *r, *fp, *temp;
	char c;
	char textoExtraido [1000],nombre_funcion[1000];
	char file1[200], file2[200], file3[200];
	char cadena[1000],cad[1000];
	temp = tmpfile();
	char ruta_fuentes[256] = "usr/lib/knn/Knn/Gpu/Fuentes/";
	char ruta_menu[256] = "usr/lib/knn/Knn/Gpu/Menus/";
	char archivo_nuevo[256];
	char palabra[1000] = "->Funcion<-";
	char argumentos[1000] = "<<< N_BLOQUES, T_per_BLOCK>>> (Elems, (int)pitch, HEAPS_dev, (int)pitch_H, QUERY_dev, (int)pitch_Q, arr_Dist, (int)pitch_Dist, Q*cont, res_final);";
	char llamada_funcion[256];
	strcpy(llamada_funcion,"Batch_Heap_Reduction<<< N_BLOQUES, T_per_BLOCK>>> (Elems, (int)pitch, HEAPS_dev, (int)pitch_H, QUERY_dev, (int)pitch_Q, arr_Dist, (int)pitch_Dist, Q*cont, res_final);");
	char cade[256];


	sprintf(archivo_nuevo,"%s%s",ruta_fuentes,argv[3]);
	printf("Ubicacion %s\n",archivo_nuevo );


	sprintf(file1, "%s", argv[1]);
	printf("\nAbriendo %s... ", argv[1]);
	fflush(stdout);
	p = fopen(file1, "r+");
	printf("OK\n");

	sprintf(file2, "%s", argv[2]);
	printf("\nAbriendo %s... ", argv[2]);
	fflush(stdout);
	q = fopen(file2, "r+");
	printf("OK\n");


	sprintf(file3, "%s", archivo_nuevo);
	printf("\nCreando %s... ", argv[3]);
	fflush(stdout);
	r = fopen(file3, "w");
	printf("OK\n");

	fflush(stdout);

	while(!feof(p))
	{
		if (fgets(cadena, 1000, p) != NULL)
		{
			if (strcmp(cadena,palabra)==10)
			{
				//fprintf(temp,"%s" , cadena);
				fprintf(temp, "%s", cadena);
				for (int i = 0; i < 1000; ++i)
				{
					fprintf(temp, " ");
				}
				//printf("encontre\n");  
			}
			else{
				fprintf(temp, "%s", cadena);
			}
		}
	}
	fprintf(temp, "\n");
	fprintf(temp, "\n");

	while(!feof(q))
	{
		fscanf(q, "%c", &c);
		fprintf(temp, "%c", c);
	}

	rewind(q);
	if (fgets(textoExtraido, 256, q)!= NULL)

		rewind(temp);

	fseek( temp, 1862, SEEK_SET );
	int len = strlen(textoExtraido);
	textoExtraido[len-1]=';';
	//printf("%d\n", len);
	fprintf( temp,"%s",textoExtraido);

	int largo = 0, aux = 0,var=0;
	while (textoExtraido[largo]!='\0'){
		if(textoExtraido[largo]==95){
			aux++;
		}
		largo++;
		if(aux==4){
			while(textoExtraido[largo] != 40){
				cad[var] = textoExtraido[largo];
				var++;
				largo++;
			}
			if (textoExtraido[largo]==40)
			{
				break;
			}
		}
		if(aux > 4){
			break;
		}
	}

	largo = 0,aux=0, var = 0;
	while (cad[largo]!='\0'){
		if(cad[largo]==32){
			aux++;
		}
		largo++;
		if(aux==2){
			while(cad[largo] != 32){
				nombre_funcion[var] = cad[largo];
				var++;
				largo++;
			}
			if (cad[largo]==32)
			{
				break;
			}
		}
		if(aux > 2){
			break;
		}
	}

	printf("NOMBRE: %s\n",nombre_funcion);
	strcat(nombre_funcion,argumentos);

	printf("FUNCION: %s\n",nombre_funcion);

	fclose(p);
	fclose(q);

	rewind(temp);

	while (!feof(temp)){
		if (fgets(cade, 256, temp) != NULL)
		{
			//printf("%d\n",strcmp(cade,llamada_funcion));
			if (strcmp(cade,llamada_funcion)==10)
			{
				printf("%s\n",nombre_funcion);
				fprintf(r, "%s\n",nombre_funcion);  
			}
			else{
				fprintf(r, "%s", cade);
			}
		}
	}

	fclose(temp);
	fclose(r);

	int existe = existsFile(archivo_nuevo);
	if (existe == 1){
		char comando[256];
		sprintf(comando,"ls %s",ruta_fuentes);
		system(comando);
		char programa[500];
		sprintf(programa, "nvcc %s -arch=sm_%d,%d -o %s", argv[3],major,minor,argv[4]);
		printf("%s\n",programa);
		system(programa);
		int executable = existsFile(argv[4]);
		if (executable){
			char fuentes[256];
			char menus[256]
			sprintf(fuentes,"echo \"%s\" >> %sfuentes.dat",argv[5],ruta_fuentes);
			sprintf(fuentes,"echo \"%s\" >> %snombres.dat",argv[5],ruta_fuentes);
			system();
		}
		
	}else{
		printf("ERROR\n");
	}

	

	

	return 0; 
}