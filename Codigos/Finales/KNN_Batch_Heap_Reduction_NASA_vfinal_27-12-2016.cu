#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <float.h>
//para el calculo de tiempos
#include <sys/resource.h>
#include <time.h>
#include <sys/time.h>

//Libreria con valores
#include "libreria.h"

#define DIM DEFINE_DIMENSION

#define T_x_BLOCK devProp_maximumThreadsPerBlock 

#define ERROR -1

#define TOPK DEFINE_TOPK

#define NE DEFINE_N_ELEM

#define TAM_WARP 32 //Num de threads maximo de un warp


/* El valor Q es la cantidad de consultas lanzadas en un kernel. Q depende de la cantidad de memoria en la GPU */
#define Q 3999

FILE *Salida;

struct _Elem
{
  double dist;
  int ind;
};
typedef struct _Elem Elem;

__device__ void insertaH(Elem *heap, Elem *elem, int *n_elem, int pitch, int id);
__device__ void extraeH(Elem *heap, int *n_elem, int pitch, int id, Elem *eresult);
__device__ double topH(Elem *heap, int id);
__device__ void popush(Elem *heap, Elem *elem, int *n_elem, int pitch, int id);
__global__ void Batch_Heap_Reduction(double *DB_dev, int pitch_DB, Elem *heap, int pitch_H, double *QUERY_dev, int pitch_QUERY, Elem *arr_Dist, int pitch_Dist, int beginQ, double *res_final);
__device__ double distancia_trans(double *p1, int pitch_p1, int col_1, double *q);
void imprime_trans(double **MAT, int col);
int leedato(double *dato, FILE *file);
int leedato_cophir(double *dato, FILE *file);
int leedato_trans(double **dato, FILE *file, int col);
int leedato_trans_cophir(double **dato, FILE *file, int col);


int N_QUERIES;
//double vectores[DIM][NE];


main(int argc, char *argv[]){

   int i, N_ELEM, dimension, j;
   FILE *pf;
   double **vectores;
   struct rusage r1, r2;
   double user_time, sys_time, real_time;
   struct timeval t1, t2;
   double *Elems, *QUERY_dev;
   double **consultas, *res_final, *res_final_H;
   int retorno, T_per_BLOCK, N_BLOQUES, contQ, cont;
   Elem *HEAPS_dev, *arr_Dist;
   size_t pitch, pitch_H, pitch_Q, pitch_Dist;  
   double prom, prom_cont;
   char path[256] = DEFINE_PATH;

//   hipSetDevice(1);
  printf("#define DEFINE_TOPK     %d\n",  DEFINE_TOPK);
  printf("#define TOPK     %d\n",  TOPK);

  if (argc != 1){
    printf("\nEjecutar como: a.out archivo_BD archivo_queries N_ELEM N_QUERIES DIM\n");
    return 0;
  }

  if ((pf = fopen(DEFINE_archivo_BD, "r")) == NULL){
    printf("\nNo se pudo abrir el archivo %s\n" ,DEFINE_archivo_BD);
    return 0;
  }



  N_ELEM = DEFINE_N_ELEM;
  N_QUERIES = DEFINE_N_QUERIES;
  dimension = DEFINE_DIMENSION;

  if (dimension != DIM ){
    printf("\nERROR :: dimension != DIM\n");
    return 0;
  }

  printf("\nCant. Elementos=%d :: dimension=%d\n" , N_ELEM, dimension);
  fflush(stdout);

  if (N_ELEM != NE){
    printf("\nERORR :: N_ELEM != NE\n");
    return 0;
  }

  if (TOPK > N_ELEM){  
    printf("ERROR  :: TOPK muy grande debe ser menor de numero de elementos de la base de datos\n");
  }

  if (T_x_BLOCK > N_ELEM)
    T_per_BLOCK = N_ELEM;
  else
    T_per_BLOCK = T_x_BLOCK;

  if (hipSuccess != hipMalloc((void **)&res_final, sizeof(double)*Q*TOPK)){
    printf("\nERROR 21 :: hipMalloc\n");
    hipDeviceReset();
    return 0;
  }

  res_final_H = (double *)malloc(sizeof(double)*Q*TOPK);
   
  for (i=0; i<Q*TOPK; i++){
    res_final_H[i] = 0;
  }

  if (hipSuccess != hipMemset(res_final, 0, sizeof(double)*Q*TOPK)){
    printf("\nERROR :: hipMemset\n");
    hipDeviceReset();
    return 0;
  }

   //HEAPS_dev[TOPK][Q*T_per_BLOCK]
  if (hipSuccess != hipMallocPitch((void **)&HEAPS_dev, &pitch_H, sizeof(Elem)*Q*T_per_BLOCK, (size_t)TOPK)){
    printf("\nERROR 4 :: hipMallocPitch :: Heaps_dev col=%lld :: row=%d\n", (long long)(sizeof(Elem)*Q*T_per_BLOCK), TOPK);
    hipDeviceReset();
    return 0;
  }

  Elem *linea_temp = (Elem *)malloc(sizeof(Elem)*Q*T_per_BLOCK);
  
  for (i=0 ; i < Q*T_per_BLOCK; i++){
    linea_temp[i].ind  = -1;
    linea_temp[i].dist = DBL_MAX; //DBL_MAX es el maximo valor para un double segun float.h
  }

  for (i=0 ; i < TOPK; i++)
    if (hipSuccess != hipMemcpy((Elem *)((char *)HEAPS_dev + (i*(int)pitch_H)), (Elem *)linea_temp, sizeof(Elem)*Q*T_per_BLOCK, hipMemcpyHostToDevice)){
      printf("\nERROR :: hipMemcpy\n");
      hipDeviceReset();
      return 0;
    }

   //arr_Dist[Q][N_ELEM]
  if (hipSuccess != hipMallocPitch((void **)&arr_Dist, &pitch_Dist, N_ELEM*sizeof(Elem), (size_t)Q)){
    printf("\nERROR 41 :: hipMallocPitch\n");
    hipDeviceReset();
    return 0;
  }

  vectores =(double **)malloc(sizeof(double *)*dimension);
  
  for (i=0; i<dimension; i++)
    vectores[i] = (double *)malloc(sizeof(double)*N_ELEM);

  for (i=0; i<N_ELEM; i++){
    for (j=0; j<dimension; j++){
      fscanf(pf, "%lf", &vectores[j][i]);
    }
      fgetc(pf);
  }
  fclose(pf);

  if (hipSuccess != hipMallocPitch((void **)&Elems, (size_t *)&pitch, N_ELEM*sizeof(double), (size_t)dimension))
    printf("\nERROR :: hipMallocPitch 4\n");

  for (i=0; i < dimension; i++){
    retorno = hipMemcpy((double *)((char *)Elems + (i*(int)pitch)), (double *)(vectores[i]), sizeof(double)*N_ELEM, hipMemcpyHostToDevice);
    if (retorno != hipSuccess){
      switch(retorno){
        case hipErrorInvalidPitchValue:
          printf("\nERROR 2 -> hipErrorInvalidPitchValue:\n");
          break;
        case hipErrorInvalidDevicePointer:
          printf("\nERROR 2 -> hipErrorInvalidDevicePointer:\n");
          break;
        case hipErrorInvalidMemcpyDirection:
          printf("\nERROR 2 -> hipErrorInvalidMemcpyDirection:\n");
          break;
        case hipErrorInvalidValue:
          printf("\nERROR 2 -> hipErrorInvalidValue :: i=%d :: pitch=%d\n", i, pitch);
          break;
        default: 
          printf("\nERROR 2 -> Checkear esto.\n");
          break;
      }
      return 0;
    }
  }

  consultas =(double **)malloc(sizeof(double *)*N_QUERIES);
   
  for (i=0; i<N_QUERIES; i++)
    consultas[i] = (double *)malloc(sizeof(double)*dimension);

  //Leo las queries
  if ((pf = fopen(DEFINE_archivo_queries, "r")) == NULL){
    printf("\nNo se pudo abrir el archivo %s\n" ,DEFINE_archivo_queries);
    return 0;
  }

  printf("\n\nArchivo de Queries:\nCant. Elementos=%d :: dimension=%d\n" , N_QUERIES, dimension);

  for (i=0; i<N_QUERIES; i++){
    if (leedato(consultas[i], pf) == -1){
      printf("\nError al leer Consultas\n");
      hipDeviceReset();
      return 0;
    }
  }
  
  fclose(pf);

   //QUERY_dev[N_QUERIES][dimension]
  if (hipSuccess != hipMallocPitch((void **)&QUERY_dev, (size_t *)&pitch_Q, dimension*sizeof(double), (size_t)N_QUERIES))
    printf("\nERROR :: hipMallocPitch 1\n");

  for (i=0; i < N_QUERIES; i++){
    if (hipSuccess != hipMemcpy((char *)QUERY_dev + (i*(int)pitch_Q), consultas[i], sizeof(double)*dimension, hipMemcpyHostToDevice))
      printf("\nERROR 3 :: hipMemcpy\n");
  }

  N_BLOQUES = Q;
  contQ = 0;
  cont = 0;
  getrusage(RUSAGE_SELF, &r1);
  gettimeofday(&t1, 0);

  while(contQ < N_QUERIES){
    contQ += Q;
    if (contQ > N_QUERIES)
    N_BLOQUES = N_QUERIES - (contQ-Q);
    printf("\nN_BLOQUES = %d :: T_per_BLOCK = %d\n", N_BLOQUES, T_per_BLOCK);
    
    Batch_Heap_Reduction<<<N_BLOQUES, T_per_BLOCK>>> (Elems, (int)pitch, HEAPS_dev, (int)pitch_H, QUERY_dev, (int)pitch_Q, arr_Dist, (int)pitch_Dist, Q*cont, res_final);
    
    if (hipSuccess != hipMemcpy((double *)res_final_H, (double *)res_final, sizeof(double)*Q*TOPK, hipMemcpyDeviceToHost)){  
      printf("\nERROR 41 :: hipMemcpy :: iteraH\n");
      hipDeviceReset();
      return 0;
    }
    cont++;
  }

  gettimeofday(&t2, 0);
  getrusage(RUSAGE_SELF, &r2);

  for (i=0; i<N_BLOQUES; i++)
  {
    fprintf(Salida,"\n\nResults array %d (smallest distances):", i);  
    for (j=TOPK*i; j<(TOPK*i)+TOPK; j++)
      fprintf(Salida,"\nquery = %d :: dist = %lf", i, res_final_H[j]);
  }
  fprintf(Salida,"\n");

  user_time = (r2.ru_utime.tv_sec - r1.ru_utime.tv_sec) + (r2.ru_utime.tv_usec - r1.ru_utime.tv_usec)/1000000.0;
  sys_time = (r2.ru_stime.tv_sec - r1.ru_stime.tv_sec) + (r2.ru_stime.tv_usec - r1.ru_stime.tv_usec)/1000000.0;
  real_time = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000;

  prom = 0;
  prom_cont = 0;
  for (i=0; i<Q; i++){
    if (res_final_H[i] != 0){
      prom += res_final_H[i];
      prom_cont += 1;
    }
  }

  printf("\nK = %d", TOPK);
  printf("\nTiempo CPU = %f", user_time + sys_time);
  printf("\nTiempo Real = %f", real_time);
  printf("\nprom = %lf\n", (double)(prom/(double)prom_cont));
  fflush(stdout);

  hipFree(Elems);
  hipFree(QUERY_dev);
  hipFree(HEAPS_dev);
  hipFree(arr_Dist);

  hipDeviceReset();
  return 0;
}


__device__ void insertaH(Elem *heap, Elem *elem, int *n_elem, int pitch, int id){
  int i;
  Elem temp;

  ((Elem *)((char *)heap + (*n_elem)*pitch))[id].dist = elem->dist;
  ((Elem *)((char *)heap + (*n_elem)*pitch))[id].ind = elem->ind;
  (*n_elem)++;
  
  for (i = *n_elem; i>1 && ((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist; i=i/2){
    //Intercambiamos con el padre
    temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
    temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist = temp.dist;
    ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].ind = temp.ind;
  }
  return;
}

__device__ void extraeH(Elem *heap, int *n_elem, int pitch, int id, Elem *eresult){
    
    int i, k;
    Elem temp;
    eresult->dist = ((Elem *)((char *)heap+0))[id].dist; //Se guarda el maximo
    eresult->ind = ((Elem *)((char *)heap+0))[id].ind; 

    ((Elem *)((char *)heap+0))[id].dist = ((Elem *)((char *)heap + ((*n_elem)-1)*pitch))[id].dist;// Movemos el ultimo a la raiz y achicamos el heap
    ((Elem *)((char *)heap+0))[id].ind = ((Elem *)((char *)heap + ((*n_elem)-1)*pitch))[id].ind;
    (*n_elem)--;
    i = 1;
    while(2*i <= *n_elem) // mientras tenga algun hijo
    {
      k = 2*i; //el hijo izquierdo
      if(k+1 <= *n_elem && ((Elem *)((char *)heap + ((k+1)-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
        k = k+1;  //el hijo derecho es el mayor

      if(((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
        break;  //es mayor que ambos hijos

      temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
      temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
      ((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + (k-1)*pitch))[id].dist;
      ((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + (k-1)*pitch))[id].ind;
      ((Elem *)((char *)heap + (k-1)*pitch))[id].dist = temp.dist;
      ((Elem *)((char *)heap + (k-1)*pitch))[id].ind = temp.ind;
      i = k;   //lo intercambiamos con el mayor hijo
    }
    return;
}

__device__ double topH(Elem *heap, int id){
  return ((Elem *)((char *)heap + 0))[id].dist;
}

__device__ void popush(Elem *heap, Elem *elem, int *n_elem, int pitch, int id){

  int i, k;
  Elem temp;

  ((Elem *)((char *)heap+0))[id].dist = elem->dist;
  ((Elem *)((char *)heap+0))[id].ind  = elem->ind;

  i = 1;
  while(2*i <= *n_elem) // mientras tenga algun hijo
  {
    k = 2*i; //el hijo izquierdo
    if(k+1 <= *n_elem && ((Elem *)((char *)heap + ((k+1)-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
      k = k+1;  //el hijo derecho es el mayor

    if(((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
      break;  //es mayor que ambos hijos

    temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
    temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + (k-1)*pitch))[id].dist;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + (k-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + (k-1)*pitch))[id].dist = temp.dist;
    ((Elem *)((char *)heap + (k-1)*pitch))[id].ind = temp.ind;
    i = k;   //lo intercambiamos con el mayor hijo
  }
  return;
}

__device__ double distancia_trans(double *p1, int pitch_p1, int col_1, double *q){

   int i=0;
   double suma=0;

   for (i=0; i < DIM; i++)
      suma += (((double *)((char *)p1 + (i*pitch_p1)))[col_1] - q[i]) * 
              (((double *)((char *)p1 + (i*pitch_p1)))[col_1] - q[i]);

   return sqrtf(suma);  
}

void imprime_trans(double **MAT, int col){
   int i;
   for (i=0; i<DIM; i++)
      printf("%lf ", MAT[i][col]);
   return;
}

int leedato(double *dato, FILE *file){
  int i=0;
   
  for (i=0;i<DIM;i++)
    if (fscanf(file,"%lf",&dato[i])<1)
      return -1;
  return 1;
}

int leedato_cophir(double *dato, FILE *file){

  int i=0;
  int num_f;
   
  for (i=0;i<DIM;i++){
    if (fscanf(file, "%d", &num_f) < 1)
      return ERROR;
    
    dato[i] = (double)num_f;
      
    if (i+1 < DIM)
      if (fgetc(file) != ','){
        printf("\nERROR :: ',' no encontrada\n");
        return ERROR;
      }
  }

  return 1;
}

int leedato_trans(double **dato, FILE *file, int col){

  int i=0;
   
  for (i=0;i<DIM;i++)
    if (fscanf(file,"%lf",&(dato[i][col]))<1)
      return -1;
  
  return 1;
}

int leedato_trans_cophir(double **dato, FILE *file, int col){

  int i=0;
  int num_f;
   
  for (i=0;i<DIM;i++){
    if (fscanf(file, "%d", &num_f) < 1)
      return ERROR;
  
    dato[i][col] = (double)num_f;
      
    if (i+1 < DIM)
      if (fgetc(file) != ','){
        printf("\nERROR :: ',' no encontrada\n");
        return ERROR;
      }
   }
   
  return 1;
}

__global__ void Batch_Heap_Reduction(double *DB_dev, int pitch_DB, Elem *heap, int pitch_H, double *QUERY_dev, int pitch_QUERY, Elem *arr_Dist, int pitch_Dist, int beginQ, double *res_final){
  int i, j, n_elem=0, n_elemWarp=0;
  int id;
  Elem eresult;
  __shared__ Elem matrizWarp[TOPK][TAM_WARP];
  __shared__ Elem heapfin[TOPK][1];
  __shared__ double query[DIM];

  id = threadIdx.x + (blockDim.x * blockIdx.x);

  //Se copia la Query a mem. compartida
  for (i=threadIdx.x; i < DIM; i += blockDim.x)
    query[i] = ((double *)((char *)QUERY_dev + ((blockIdx.x + beginQ) * (int)pitch_QUERY)))[i];

  __syncthreads();

  //Se obtiene el arreglo de distancias
  for (i=threadIdx.x; i < NE; i += blockDim.x)
  {
    ((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i].dist = distancia_trans(DB_dev, pitch_DB, i, query);
    ((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i].ind = i;
  }

  for(i=threadIdx.x; i < NE; i += blockDim.x){
    //NE = Numero de elementos de la BD
    if (n_elem >= TOPK){
      if (topH(heap, id) > ((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i].dist)
        popush(heap, &(((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i]), &n_elem, pitch_H, id); //Extrae e inserta en una operacion
      }
    else
      insertaH(heap, &(((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i]), &n_elem, pitch_H, id);
  }
  __syncthreads();


    //Un warp reduce el problema a una matriz de Kx32 distancias. PEro esta vez los heaps se almacenan en Memoria Shared
  if (threadIdx.x < TAM_WARP){
    for(j=id; j < blockDim.x*(blockIdx.x+1); j += TAM_WARP){
      n_elem = TOPK;
      for(i=0; i < TOPK; i++){
        extraeH(heap, &n_elem, pitch_H, j, &eresult);
        if (n_elemWarp < TOPK)
          insertaH(&(matrizWarp[0][0]), &eresult, &n_elemWarp, sizeof(Elem)*TAM_WARP, threadIdx.x);
        else
          if (topH(&(matrizWarp[0][0]), threadIdx.x) > eresult.dist)
            popush(&(matrizWarp[0][0]), &eresult, &n_elemWarp, sizeof(Elem)*TAM_WARP, threadIdx.x);
      }
    }
  }
  __syncthreads();

  //Un hilo encuentra los K-NN a partir de la matriz de TOPKxTAM_WARP
  if (threadIdx.x == 0){
    n_elem = 0;
    for(j=0; j < TAM_WARP; j++){
      for(i=0; i < TOPK; i++)
        if (n_elem < TOPK)
          insertaH((Elem *)heapfin, &(matrizWarp[i][j]), &n_elem, sizeof(Elem), 0);
        else
          if (topH((Elem *)heapfin, 0) > matrizWarp[i][j].dist)
            popush((Elem *)heapfin, &(matrizWarp[i][j]), &n_elem, sizeof(Elem), 0);
    }

    //Escribiendo algunos resultados
    //res_final[blockIdx.x] = topH((Elem *)heapfin, 0);

    for (i=TOPK*blockIdx.x; i < (TOPK*blockIdx.x)+TOPK; i++){
      extraeH(&(heapfin[0][0]), &n_elem, sizeof(Elem), 0, &eresult);
      res_final[i] = eresult.dist;
    }
  }

  //atomicAdd(&(resT[blockIdx.x]), ED);
  return;
}
