#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 10
#define M 20

__global__ void matrix_add(int *matrix_A, size_t pitch_A, int *matrix_B, size_t pitch_B, int *matrix_C, size_t pitch_C)
{
	int row, col;
	row = threadIdx.x;
	col = threadIdx.y;
	for (int i = 0; i < row; ++i){
		for (int j = 0; j < col; ++j){
			matrix_C[i][j]=matrix_A[i][j]+matrix_B[i][j];
		}	
	}
	
}


int main()
{
	int a[N][M], b[N][M], c[N][M], i, j;
	int *matrix_C, *matrix_B, *matrix_A;
	size_t pitch_A, pitch_B, pitch_C;


	/* allocate space for device copies of a, b, c */
	/* the pitch_A, pitch_B and pitch_C are the assigned sizes of a row in the matrices matrix_A, matrix_B and matrix_C respectively */
	if (hipSuccess != hipMallocPitch((void **)&matrix_A, &pitch_A, N * sizeof(int), M))
	{
		printf("\nERROR :: hipMallocPitch :: matrix_A\n");
		hipDeviceReset();
		return 0;
	}

	if (hipSuccess != hipMallocPitch((void **)&matrix_B, &pitch_B, N * sizeof(int), M))
	{
		printf("\nERROR :: hipMallocPitch :: matrix_B\n");
		hipDeviceReset();
		return 0;
	}

	if (hipSuccess != hipMallocPitch((void **)&matrix_C, &pitch_C, N * sizeof(int), M))
	{
		printf("\nERROR :: hipMallocPitch :: matrix_C\n");
		hipDeviceReset();
		return 0;
	}

	for( i = 0; i < N; i++ )
		for( j = 0; j < M; j++ ){
			a[i][j] = 1;
			b[i][j] = 10;
			c[i][j] = 0;
		}

	printf( "\nMatriz A:\n\n");
	for( i = 0; i < N; i++ ){
		for( j = 0; j < M; j++ )
			printf( "%d ", a[i][j] );
		printf( "\n");
	}

	printf( "\nMatriz B:\n\n");
	for( i = 0; i < N; i++ ){
		for( j = 0; j < M; j++ )
			printf( "%d ", b[i][j] );
		printf( "\n");
	}

	//Copying from host to device the 2D-matrices a and b
	hipMemcpy2D(matrix_A, pitch_A, *a, sizeof(int)*N, sizeof(int)*N, M, hipMemcpyHostToDevice);
	hipMemcpy2D(matrix_B, pitch_B, *b, sizeof(int)*N, sizeof(int)*N, M, hipMemcpyHostToDevice);

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Record the start event
	hipEventRecord(start, NULL);


	/* launch the kernel on the GPU with 1 CUDA Block and N threads per CUDA Block*/
	matrix_add<<< 1, M >>>( matrix_A, pitch_A, matrix_B, pitch_B, matrix_C, pitch_C);


	// Record the stop event
	hipEventRecord(stop, NULL);

	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);

	printf("Running Time (msecs.) = %f\n", msecTotal);

	/* copy result back to host */
	//Copying from host to device the 2D-matrices a and b
	hipMemcpy2D(*c, sizeof(int)*N, matrix_C, pitch_C, sizeof(int)*N, M, hipMemcpyDeviceToHost);

	printf( "\nMatriz C:\n\n");
	for( i = 0; i < N; i++ ){
		for( j = 0; j < M; j++ )
			printf( "%d ", c[i][j] );
		printf( "\n");
	}
	
	/* clean up */
	hipFree( matrix_A );
	hipFree( matrix_B );
	hipFree( matrix_C );
	
	return 0;
} /* end main */



