#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>

//To measure time:
#include <sys/resource.h>
#include <time.h>
#include <sys/time.h>

#define ERROR -1
#define LENGTH_ARRAY 100000 
#define TOPK 32 
#define TAM_WARP 32 
#define N_BLOQUES 3 
#define T_per_BLOCK 512 

// Global variables
int DIM;
FILE *Salida_Multihilo;

struct _Elem{
	float dist;
	int ind;
};
typedef struct _Elem Elem;


void copiavalor(double *a, double *b);
int leedato(double *dato, FILE *file);
__device__ void pushH(Elem *heap, Elem *elem, int *n_elem, int pitch, int id);
__device__ void popH(Elem *heap, int *n_elem, int pitch, int id, Elem *eresult);
__device__ float topH(Elem *heap, int id);
__device__ void popushH(Elem *heap, Elem *elem, int *n_elem, int pitch, int id);
__global__ void Batch_Heap_Reduction(Elem *heap, int pitch_H, Elem *arr_Dist, int pitch_Dist, Elem *res_final);


main(int argc, char *argv[])
{
	int i, j,N_QUERIES,N_DB;
	
	char *ruta_db, *ruta_queries;
	double **DB, **Consultas,*CudaDB, *CudaConsultas;
	size_t pitch_DB, pitch_Consultas;

	
	if (argc != 6){
		printf("Error :: Ejecutar como : main.out archivo_BD Num_elem archivo_queries Num_queries dim\n");
		return 1;
	}
	
	ruta_db = (char *)malloc(sizeof(char)*(strlen(argv[1])+1));
	strcpy(ruta_db, argv[1]);
	N_DB = atoi(argv[2]);

	printf("%s\n",ruta_db );

	ruta_queries = (char *)malloc(sizeof(char)*(strlen(argv[3])+1));
	strcpy(ruta_queries, argv[3]);
	N_QUERIES = atoi(argv[4]);
	printf("N_QUERIES:: -> :: %d\n",N_QUERIES );
	DIM = atoi(argv[5]);

	printf("dim:: %d\n",DIM );

	Elem *res_final, *res_final_H;
	Elem *HEAPS_dev, *arr_Dist, **arr_Dist_H;
	size_t pitch_H, pitch_Dist;
	//Variable for time:
	struct rusage r1, r2;
	float user_time, sys_time, real_time;
	struct timeval t1, t2;
	FILE *f_dist, *fquery;
	double dato[DIM],datos[DIM];

	//Lectura de Base de datos
	
	if (hipSuccess != hipMallocPitch((void **)&CudaDB, &pitch_DB, DIM * sizeof(int), N_DB)){
		printf("\nERROR :: hipMallocPitch :: CudaDB\n");
		hipDeviceReset();
		return 0;
	}

	if (hipSuccess != hipMallocPitch((void **)&CudaConsultas, &pitch_Consultas, DIM * sizeof(int), N_QUERIES)){
		printf("\nERROR :: hipMallocPitch :: CudaConsultas\n");
		hipDeviceReset();
		return 0;
	}

	printf("\nAbriendo %s... ", argv[1]);
	fflush(stdout);
	f_dist = fopen(ruta_db, "r");
	printf("OK\n");
	fflush(stdout);

	Consultas = (double **) malloc(sizeof (double *)*N_QUERIES);
	for (i = 0; i < N_QUERIES; i++)
		Consultas[i] = (double *) malloc(sizeof (double)*DIM);

	DB = (double **) malloc(sizeof (double *)*N_DB);
	for (i = 0; i < N_DB; i++)
		DB[i] = (double *) malloc(sizeof (double)*DIM);

	printf("\nCargando DB... ");
	fflush(stdout);
	for (i = 0; i < N_DB; i++) {
		if (leedato(dato, f_dist) == ERROR || feof(f_dist)) {
			printf("\n\nERROR :: N_DB mal establecido\n\n");
			fflush(stdout);
			fclose(f_dist);
			break;
		}
		copiavalor(DB[i], dato);
	}
	fclose(f_dist);
	printf("OK\n");
	fflush(stdout);

	if ((fquery = fopen(ruta_queries, "r")) == NULL)
		printf("Error al abrir para lectura el archivo de queries: %s\n", ruta_queries);
	else
		printf("Abriendo  para lectura %s\n", ruta_queries);
	printf("\nCargando Consultas... ");
	fflush(stdout);
	for (i = 0; i < N_QUERIES;   i++) {
		if (leedato(datos, fquery) == ERROR || feof(fquery)) {
			printf("\n\nERROR :: N_QUERIES mal establecido, Menos queries que las indicadas\n\n");
			fflush(stdout);
			fclose(fquery);
			break;
		}
		copiavalor(Consultas[i], datos);
	}
	fclose(fquery);
	printf("OK\n");
	fflush(stdout);

	printf("\nLength of the arrays = %d" , LENGTH_ARRAY);
	printf("\nTOPK = %d" , TOPK);
	fflush(stdout);

	for( i = 0; i < dim; i++ ){
		for( j = 0; j < n_db; j++ )
		{
			CudaDB[j][i] = DB[i][j];
		}
	}

	//Allocating space to store the results
	if (hipSuccess != hipMalloc((void **)&res_final, sizeof(Elem)*TOPK*N_BLOQUES))
	{
		printf("\nERROR 1 :: hipMalloc\n");
		hipDeviceReset();
		return 0;
	}
	res_final_H = (Elem *)malloc(sizeof(Elem)*TOPK*N_BLOQUES);
	for (i=0; i<TOPK*N_BLOQUES; i++)
	{
		res_final_H[i].ind = 0;
		res_final_H[i].dist = 0;
	}
	//Initializing res_final
	if (hipSuccess != hipMemset(res_final, 0, sizeof(Elem)*TOPK*N_BLOQUES))
	{
		printf("\nERROR 2 :: hipMemset\n");
		hipDeviceReset();
		return 0;
	}

	 //Allocating space for the heaps in HEAPS_dev[TOPK][N_BLOQUES*512]. The elements of the heaps are accesed by columns to improve coalescing.
	if (hipSuccess != hipMallocPitch((void **)&HEAPS_dev, &pitch_H, sizeof(Elem)*N_BLOQUES*T_per_BLOCK, TOPK))
	{
		printf("\nERROR 3 :: hipMallocPitch :: Heaps_dev col=%lld :: row=%d\n", (long long)(sizeof(Elem)*N_BLOQUES*T_per_BLOCK), TOPK);
		hipDeviceReset();
		return 0;
	}

	//Generating the arrays to be reduced
	arr_Dist_H = (Elem **)malloc(sizeof(Elem *)*N_BLOQUES);
	for (i=0; i<N_BLOQUES; i++)
		arr_Dist_H[i] = (Elem *)malloc(sizeof(Elem)*LENGTH_ARRAY);

	for (i=0; i<N_BLOQUES; i++)
		for (j=0; j<LENGTH_ARRAY; j++)
		{
			arr_Dist_H[i][j].ind = (LENGTH_ARRAY*i) + j; //Setting an ID
			arr_Dist_H[i][j].dist = (float)(0.1*(float)((LENGTH_ARRAY * i) + j));//Setting as distances the numbers 0, 0.1, 0.2, ...
		}

	 //Allocating space for the arrays to be reduced in arr_Dist[N_BLOQUES][LENGTH_ARRAY]. One array per row.
		if (hipSuccess != hipMallocPitch((void **)&arr_Dist, &pitch_Dist, LENGTH_ARRAY*sizeof(Elem), N_BLOQUES))
		{
			printf("\nERROR 4 :: hipMallocPitch\n");
			hipDeviceReset();
			return 0;
		}

	//Copying the arrays to be reduced from host to device memory
		for (i=0; i < N_BLOQUES; i++)
			if (hipSuccess != hipMemcpy((char *)arr_Dist + (i*(int)pitch_Dist), (Elem *)(arr_Dist_H[i]), sizeof(Elem)*LENGTH_ARRAY, hipMemcpyHostToDevice))
			{
				printf("\nERROR 5 :: hipMemcpy\n");
				hipDeviceReset();
				return 0;
			}

	 //We finish with all the unresolved operations
			hipDeviceSynchronize();
			hipDeviceSynchronize();
	//Measuring time
			getrusage(RUSAGE_SELF, &r1);
			gettimeofday(&t1, 0);

			printf("\nN_BLOQUES = %d :: T_per_BLOCK = %d\n", N_BLOQUES, T_per_BLOCK);
			fflush(stdout);

			Batch_Heap_Reduction<<< N_BLOQUES, T_per_BLOCK>>> (HEAPS_dev, (int)pitch_H, arr_Dist, (int)pitch_Dist, res_final);

			if (hipSuccess != hipMemcpy((Elem *)res_final_H, (Elem *)res_final, sizeof(Elem)*TOPK*N_BLOQUES, hipMemcpyDeviceToHost))
			{
				printf("\nERROR 41 :: hipMemcpy :: iteraH\n");
				hipDeviceReset();
				return 0;
			}

			hipDeviceSynchronize();
			hipDeviceSynchronize();
			gettimeofday(&t2, 0);
			getrusage(RUSAGE_SELF, &r2);

			user_time = (r2.ru_utime.tv_sec - r1.ru_utime.tv_sec) + (r2.ru_utime.tv_usec - r1.ru_utime.tv_usec)/1000000.0;
			sys_time = (r2.ru_stime.tv_sec - r1.ru_stime.tv_sec) + (r2.ru_stime.tv_usec - r1.ru_stime.tv_usec)/1000000.0;
			real_time = (t2.tv_sec - t1.tv_sec) + (float)(t2.tv_usec - t1.tv_usec)/1000000;

			printf("\nK = %d", TOPK);
			printf("\nTiempo CPU = %f", user_time + sys_time);
			printf("\nTiempo Real = %f", real_time);
			fflush(stdout);

			for (i=0; i<N_BLOQUES; i++)
			{
				printf("\n\nResults array %d (smallest distances):", i);  
				for (j=TOPK*i; j<(TOPK*i)+TOPK; j++)
					printf("\nind = %d :: dist = %f", res_final_H[j].ind, res_final_H[j].dist);
			}
			printf("\n");

			hipFree(HEAPS_dev);
			hipFree(arr_Dist);

			hipDeviceReset();
			return 0;
		}

//Push an element 'elem' to the id-th heap stored in the id-th column of the matrix 'heap'
		__device__ void pushH(Elem *heap, Elem *elem, int *n_elem, int pitch, int id)
		{
			int i;
			Elem temp;

			((Elem *)((char *)heap + (*n_elem)*pitch))[id].dist = elem->dist;
			((Elem *)((char *)heap + (*n_elem)*pitch))[id].ind = elem->ind;
			(*n_elem)++;
			for (i = *n_elem; i>1 && ((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist; i=i/2)
			{
		//Swap with the father
				temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
				temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
				((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist;
				((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].ind;
				((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist = temp.dist;
				((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].ind = temp.ind;
			}
			return;
		}

//Pop an element from id-th heap stored in the id-th column of the matrix 'heap' and stores it in 'eresult'
		__device__ void popH(Elem *heap, int *n_elem, int pitch, int id, Elem *eresult)
		{
			int i, k;
			Elem temp;
			eresult->dist = ((Elem *)((char *)heap+0))[id].dist;
			eresult->ind = ((Elem *)((char *)heap+0))[id].ind; 

	((Elem *)((char *)heap+0))[id].dist = ((Elem *)((char *)heap + ((*n_elem)-1)*pitch))[id].dist;//Moving the last element to the root
	((Elem *)((char *)heap+0))[id].ind = ((Elem *)((char *)heap + ((*n_elem)-1)*pitch))[id].ind;
	(*n_elem)--;
	i = 1;
	while(2*i <= *n_elem) //while exists some child
	{
		k = 2*i; //left child
		if(k+1 <= *n_elem && ((Elem *)((char *)heap + ((k+1)-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
			k = k+1;  //right child is the biggest

		if(((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
			break;  //bigger than both childs

		temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
		temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
		((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + (k-1)*pitch))[id].dist;
		((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + (k-1)*pitch))[id].ind;
		((Elem *)((char *)heap + (k-1)*pitch))[id].dist = temp.dist;
		((Elem *)((char *)heap + (k-1)*pitch))[id].ind = temp.ind;
		i = k;   //swap with the biggest child
	}
	return;
}


//Returns the root of the id-th heap (stored in the id-th column)
__device__ float topH(Elem *heap, int id) //NOTE: Be careful if the heap is empty and topH is called, it will give an error
{
	return ((Elem *)((char *)heap + 0))[id].dist;
}

//Pop and push in one operation
__device__ void popushH(Elem *heap, Elem *elem, int *n_elem, int pitch, int id)
{
	int i, k;
	Elem temp;

	((Elem *)((char *)heap+0))[id].dist = elem->dist;
	((Elem *)((char *)heap+0))[id].ind  = elem->ind;

	i = 1;
	while(2*i <= *n_elem) //while exists some child
	{
		k = 2*i; //left child
		if(k+1 <= *n_elem && ((Elem *)((char *)heap + ((k+1)-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
			k = k+1;  //right child is the biggest

		if(((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
			break;  //bigger than both childs

		temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
		temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
		((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + (k-1)*pitch))[id].dist;
		((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + (k-1)*pitch))[id].ind;
		((Elem *)((char *)heap + (k-1)*pitch))[id].dist = temp.dist;
		((Elem *)((char *)heap + (k-1)*pitch))[id].ind = temp.ind;
		i = k;   //swap with the bigger child
	}
	return;
}


__global__ void Batch_Heap_Reduction(Elem *heap, int pitch_H, Elem *arr_Dist, int pitch_Dist, Elem *res_final)
{
	int i, j, n_elem=0, n_elemWarp=0;
	int id;
	Elem eresult;
	__shared__ Elem matrizWarp[TOPK][TAM_WARP];
	__shared__ Elem heapfin[TOPK][1];

	id = threadIdx.x + (blockDim.x * blockIdx.x);

	//First Step: The array to be sorted (arr_Dist) is reduced to T_per_BLOCK heaps stored in device memory
	//The element of arr_Dist are distributed in a circular manner, therefore consecutive threads access to consecutive elements (consecutive memory addresses)
	for(i=threadIdx.x; i < LENGTH_ARRAY; i += blockDim.x)
	{
			if (n_elem >= TOPK)//If the current number of elements in the heap is >= than TOPK (really never it is > than TOPK, at most it is equal to TOPK)
			{
			//The next if is to add an element to the heap just if that element is less than the head of the heap
				if (topH(heap, id) > ((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i].dist)
						popushH(heap, &(((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i]), &n_elem, pitch_H, id); //Pop and Push in one operation
				}
				else
					pushH(heap, &(((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i]), &n_elem, pitch_H, id);
			}

			__syncthreads();


	//Second Step: the first warp of the CUDA Block reduces the elements of the heaps (stored in device memory) to TAM_WARP heaps stored in shared memory
			if (threadIdx.x < TAM_WARP)
			{
				for(j=id; j < blockDim.x*(blockIdx.x+1); j += TAM_WARP)
				{
					n_elem = TOPK;
					for(i=0; i < TOPK; i++)
					{
				 popH(heap, &n_elem, pitch_H, j, &eresult);//Getting an element from a heap in device memory

			//Adding the element to the heap in shared memory (if it corresponds)
				 if (n_elemWarp < TOPK)
				 	pushH(&(matrizWarp[0][0]), &eresult, &n_elemWarp, sizeof(Elem)*TAM_WARP, threadIdx.x);
				 else
				 	if (topH(&(matrizWarp[0][0]), threadIdx.x) > eresult.dist)
				 		popushH(&(matrizWarp[0][0]), &eresult, &n_elemWarp, sizeof(Elem)*TAM_WARP, threadIdx.x);
				 }
				}
			}


			__syncthreads();


	//Third Step: The first thread of the CUDA Block reduces the elements to one heap (stored in shared memory). The elements of this heap are the final results.
			if (threadIdx.x == 0)
			{
				n_elem = 0;
				for(j=0; j < TAM_WARP; j++)
				{
					for(i=0; i < TOPK; i++)
						if (n_elem < TOPK)
							pushH((Elem *)heapfin, &(matrizWarp[i][j]), &n_elem, sizeof(Elem), 0);
						else
							if (topH((Elem *)heapfin, 0) > matrizWarp[i][j].dist)
								popushH((Elem *)heapfin, &(matrizWarp[i][j]), &n_elem, sizeof(Elem), 0);
						}

		 //Writing the results
						for (i=TOPK*blockIdx.x; i<TOPK*(blockIdx.x+1); i++)
							popH(&(heapfin[0][0]), &n_elem, sizeof(Elem), 0, &(res_final[i]));
					}

					return;
				}

				void copiavalor(double *a, double *b) {
					int i;
					for (i = 0; i < DIM; i++)
						a[i] = b[i];
					return;
				}
				int leedato(double *dato, FILE *file) {
					int i = 0;

					for (i = 0; i < DIM; i++)
						if (fscanf(file, "%lf", &dato[i]) < 1)
							return ERROR;
						return 1;
					}